#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <iomanip>
#include <iostream>
#include <ostream>
#include <random>
#include <time.h>
#include <unordered_set>
#include <vector>
#include <mma.h>

using namespace nvcuda;

#include "benchmark/benchmark.h"

// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// #include <iostream>
// #include <benchmark/benchmark.h>

// Utility function to check CUDA errors
#define CHECK_CUDA(func)                                                       \
{                                                                             \
    hipError_t status = (func);                                             \
    if (status != hipSuccess) {                                             \
        printf("CUDA API failed at %s line %d with error: %s (%d)\n",        \
               __FILE__, __LINE__, hipGetErrorString(status), status);       \
        return;                                                 \
    }                                                                         \
}

// Utility function to check cuBLAS errors
#define CHECK_CUBLAS(func)                                                    \
{                                                                            \
    hipblasStatus_t status = (func);                                          \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                   \
        printf("cuBLAS API failed at %s line %d with error: %d\n",          \
               __FILE__, __LINE__, status);                                   \
       return;                                                  \
    }                                                                         \
}

// Utility function to check cuSPARSE errors
#define CHECK_CUSPARSE(func)                                                  \
{                                                                            \
    hipsparseStatus_t status = (func);                                        \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                 \
        printf("cuSPARSE API failed at %s line %d with error: %d\n",        \
               __FILE__, __LINE__, status);                                   \
       return;                                                   \
    }                                                                         \
}

// Function to generate random sparse matrix with given sparsity
void generate_sparse_matrix(float *matrix, int rows, int cols, float sparsity, int seed = 0) {
    srand(seed);
    auto total_nnz = (int) (rows * cols * (1 - sparsity));
    // generate random unique integer indices between 0 and rows*cols
    std::unordered_set<int> unique_numbers;
    std::vector<int> indices;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, rows * cols - 1);

    while (unique_numbers.size() < total_nnz) {
        int num = dis(gen);
        if (unique_numbers.insert(num).second) {
            indices.push_back(num);
        }
    }
    // fill the matrix with random values
    for (int i = 0; i < rows * cols; i++) {
        matrix[i] = 0.0f;
    }
    for (int i = 0; i < total_nnz; i++) {
        matrix[indices[i]] = (float) rand() / RAND_MAX;
    }
}

// Function to convert dense matrix to CSR format
void dense_to_csr(float *dense, int rows, int cols,
                  float **values, int **row_ptr, int **col_ind,
                  int *nnz) {
    // First pass: count non-zero elements
    *nnz = 0;
    for (int i = 0; i < rows * cols; i++) {
        if (dense[i] != 0.0f) {
            (*nnz)++;
        }
    }

    // Allocate memory
    *values = (float *) malloc(*nnz * sizeof(float));
    *col_ind = (int *) malloc(*nnz * sizeof(int));
    *row_ptr = (int *) malloc((rows + 1) * sizeof(int));

    // Second pass: fill CSR arrays
    int count = 0;
    (*row_ptr)[0] = 0;

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if (dense[i * cols + j] != 0.0f) {
                (*values)[count] = dense[i * cols + j];
                (*col_ind)[count] = j;
                count++;
            }
        }
        (*row_ptr)[i + 1] = count;
    }
}


__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K, float alpha, float beta) {
    // Leading dimensions. Packed with no transpositions.
    int lda = M;
    int ldb = K;
    int ldc = M;

    // Tile using a 2D grid
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // Loop over k
    for (int i = 0; i < K; i += WMMA_K) {
        int aRow = warpM * WMMA_M;
        int aCol = i;

        int bRow = i;
        int bCol = warpN * WMMA_N;

        // Bounds checking
        if (aRow < M && aCol < K && bRow < K && bCol < N) {
            // Load the inputs
            wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
            wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

        }
    }

    // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
    int cRow = warpM * WMMA_M;
    int cCol = warpN * WMMA_N;

    if (cRow < M && cCol < N) {
        wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);

#pragma unroll
        for (int i = 0; i < c_frag.num_elements; i++) {
            c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
        }

        // Store the output
        wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
    }
}

__global__ void convertFp32ToFp16(half *out, float *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx];
    }
}

void MeasureGEMMPerformance(hipblasHandle_t handle, int m, int n, int k,
                            const float *d_A, const float *d_B, float *d_C,
                            const float alpha, const float beta,
                            float &elapsed_time) {
    hipEvent_t event_start, event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);

    hipEventRecord(event_start, 0);
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             n, m, k,
                             &alpha,
                             d_B, n,
                             d_A, k,
                             &beta,
                             d_C, n));
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&elapsed_time, event_start, event_stop);

    hipEventDestroy(event_start);
    hipEventDestroy(event_stop);
}

void MeasureTensorCoreGEMMPerformance(hipblasHandle_t handle, int m, int n, int k,
                                      const float *d_A, const float *d_B, float *d_C,
                                      const float alpha, const float beta,
                                      float &elapsed_time) {
    hipEvent_t event_start, event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);

    hipblasGemmAlgo_t CuBlasALG = static_cast<hipblasGemmAlgo_t>(0);

    hipEventRecord(event_start, 0);
    CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              n, m, k,
                              &alpha,
                              d_B, HIP_R_32F, n,
                              d_A, HIP_R_32F, k,
                              &beta,
                              d_C, HIP_R_32F, n, HIP_R_32F, CuBlasALG));
    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&elapsed_time, event_start, event_stop);

    hipEventDestroy(event_start);
    hipEventDestroy(event_stop);
}

void MeasureTensorCorePerformance(int m, int n, int k, half *d_A,
                                  half *d_B, float *d_C,
                                  float alpha, float beta, float &elapsed_time) {

    hipEvent_t event_start, event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);

    // First: using WMMA
    dim3 gridDim;
    dim3 blockDim;

    // blockDim.x must be a multple of warpSize
    // 128x4 means we have 16 warps and a block computes a 64x64 output tile
    blockDim.x = 128;
    blockDim.y = 4;

    gridDim.x = (m + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
    gridDim.y = (m + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

    hipEventRecord(event_start, 0);

    wmma_example <<< gridDim, blockDim >>>(d_A, d_B, d_C, m, n, k, alpha, beta);

    hipEventRecord(event_stop, 0);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&elapsed_time, event_start, event_stop);

    hipEventDestroy(event_start);
    hipEventDestroy(event_stop);

}

// benchmark for sparse matrix multiplication
static void BM_cuBLAS_CUDA(benchmark::State &state) {
    int m = state.range(0);
    int n = state.range(1);
    int k = state.range(2);
    auto sparsity = state.range(3);
    float sparsity_ratio = sparsity / 100.0f;
    // Initialize CUDA handles
    hipblasHandle_t cublas_handle;
    CHECK_CUBLAS(hipblasCreate(&cublas_handle));

    // Allocate host memory for dense matrices
    float *h_A = (float *) malloc(m * k * sizeof(float));
    float *h_B = (float *) malloc(k * n * sizeof(float));
    float *h_C = (float *) malloc(m * n * sizeof(float));

    // Generate random sparse matrix A and dense matrix B
    generate_sparse_matrix(h_A, m, k, sparsity_ratio);
    generate_sparse_matrix(h_B, k, n, 0.0f);  // B is dense

    // Allocate device memory for dense matrices
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc((void **) &d_A, m * k * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &d_B, k * n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &d_C, m * n * sizeof(float)));

    // Constants for GEMM
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const char *device_name = prop.name;

    // add gpu name to the log
    // add sparsity to the log
    state.SetLabel("Arch:" + std::string(device_name));
    for (auto _: state) {
        float iteration_time_ms = 0.0f;
        MeasureGEMMPerformance(cublas_handle, m, n, k, d_A, d_B, d_C,
                               alpha, beta, iteration_time_ms);
        state.SetIterationTime(iteration_time_ms);


    }
    // Final cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUBLAS(hipblasDestroy(cublas_handle));
}

static void BM_cuBLAS_TensorCore(benchmark::State &state) {
    int m = state.range(0);
    int n = state.range(1);
    int k = state.range(2);
    auto sparsity = state.range(3);
    float sparsity_ratio = sparsity / 100.0f;
    // Initialize CUDA handles
    hipblasHandle_t cublas_handle;
    CHECK_CUBLAS(hipblasCreate(&cublas_handle));
    hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
    // Allocate host memory for dense matrices
    float *h_A = (float *) malloc(m * k * sizeof(float));
    float *h_B = (float *) malloc(k * n * sizeof(float));
    float *h_C = (float *) malloc(m * n * sizeof(float));

    // Generate random sparse matrix A and dense matrix B
    generate_sparse_matrix(h_A, m, k, sparsity_ratio);
    generate_sparse_matrix(h_B, k, n, 0.0f);  // B is dense

    // Allocate device memory for dense matrices
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc((void **) &d_A, m * k * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &d_B, k * n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &d_C, m * n * sizeof(float)));

    // Constants for GEMM
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const char *device_name = prop.name;

    // add gpu name to the log
    // add sparsity to the log
    state.SetLabel("Arch:" + std::string(device_name));
    for (auto _: state) {
        float iteration_time_ms = 0.0f;
        MeasureTensorCoreGEMMPerformance(cublas_handle, m, n, k, d_A, d_B, d_C,
                               alpha, beta, iteration_time_ms);
        state.SetIterationTime(iteration_time_ms);


    }
    // Final cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUBLAS(hipblasDestroy(cublas_handle));
}

static void BM_CUSPARSE_SPMM(benchmark::State &state) {
    auto m = state.range(0);
    auto n = state.range(1);
    auto k = state.range(2);
    auto sparsity = state.range(3);
    float sparsity_ratio = sparsity / 100.0f;
    // hipError_t cudaStatus = hipSetDevice(0);
    // if (cudaStatus != hipSuccess) {
    //     printf("CUDA context creation failed with error: %s\n", hipGetErrorString(cudaStatus));
    //     return;
    // }
    // Check CUDA context creation using hipFree(0)
    hipError_t cudaStatus = hipFree(0);
    if (cudaStatus != hipSuccess) {
        printf("CUDA context creation failed with error: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    // Initialize CUDA handles
    hipsparseHandle_t cusparse_handle;
    if (hipsparseCreate(&cusparse_handle) != HIPSPARSE_STATUS_SUCCESS) {
        state.SkipWithError("hipsparseCreate failed");
        return;
    }

    // Allocate host memory for dense matrices
    auto *h_A = (float *) malloc(m * k * sizeof(float));
    auto *h_B = (float *) malloc(k * n * sizeof(float));
    auto *h_C = (float *) malloc(m * n * sizeof(float));

    // Generate random sparse matrix A and dense matrix B
    generate_sparse_matrix(h_A, m, k, sparsity_ratio);
    generate_sparse_matrix(h_B, k, n, 0.0f);  // B is dense

    // Allocate device memory for dense matrices
    float *d_B, *d_C;
    CHECK_CUDA(hipMalloc((void **) &d_B, k * n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &d_C, m * n * sizeof(float)));

    // Constants for GEMM
    const float alpha = 1.0f;
    const float beta = 0.0f;
    // Convert matrix A to CSR format
    float *h_csrVal;
    int *h_csrRowPtr, *h_csrColInd;
    int nnz;
    dense_to_csr(h_A, m, k, &h_csrVal, &h_csrRowPtr, &h_csrColInd, &nnz);


    // Allocate device memory for sparse matrix
    float *d_csrVal;
    int *d_csrRowPtr, *d_csrColInd;
    CHECK_CUDA(hipMalloc((void **) &d_csrVal, nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &d_csrRowPtr, (m + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **) &d_csrColInd, nnz * sizeof(int)));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_csrVal, h_csrVal, nnz * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrRowPtr, h_csrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrColInd, h_csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice));

    // Create matrix descriptors for sparse operations
    hipsparseMatDescr_t descr;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr));
    CHECK_CUSPARSE(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));


    // Create cusparseSpMat and cusparseDnMat descriptors
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;

    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, m, k, nnz,
                                     d_csrRowPtr, d_csrColInd, d_csrVal,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, k, n, k, d_B,
                                       HIP_R_32F, HIPSPARSE_ORDER_COL));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, m, n, m, d_C,
                                       HIP_R_32F, HIPSPARSE_ORDER_COL));

    // Temporary buffer
    size_t bufferSize;
    void *dBuffer = nullptr;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(cusparse_handle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, matA, matB, &beta, matC,
                                           HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const char *device_name = prop.name;

    // add sparsity to the log
    state.SetLabel("Arch:" + std::string(device_name));

    for (auto _: state) {
        float iteration_time_ms = 0.0f;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
        // Perform SpMM

        // Perform SpMM
        CHECK_CUSPARSE(hipsparseSpMM(cusparse_handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, matA, matB, &beta, matC,
                                    HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer));
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&iteration_time_ms, start, stop);
        state.SetIterationTime(iteration_time_ms);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    // Clean up
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC));
    //CHECK_CUDA(hipFree(dBuffer));


    // Clean up sparse matrix resources
    free(h_csrVal);
    free(h_csrRowPtr);
    free(h_csrColInd);
    CHECK_CUDA(hipFree(d_csrVal));
    CHECK_CUDA(hipFree(d_csrRowPtr));
    CHECK_CUDA(hipFree(d_csrColInd));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));

    // Final cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    CHECK_CUSPARSE(hipsparseDestroy(cusparse_handle));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
}

static void BM_TensorCore_GEMM(benchmark::State &state) {
    int m = state.range(0);
    int n = state.range(1);
    int k = state.range(2);
    auto sparsity = state.range(3);
    float sparsity_ratio = sparsity / 100.0f;

    // Allocate host memory for dense matrices
    float *h_A = (float *) malloc(m * k * sizeof(float));
    float *h_B = (float *) malloc(k * n * sizeof(float));
    float *h_C = (float *) malloc(m * n * sizeof(float));

    // Generate random sparse matrix A and dense matrix B
    generate_sparse_matrix(h_A, m, k, sparsity_ratio);
    generate_sparse_matrix(h_B, k, n, 0.0f);

    // Allocate device memory for dense matrices
    float *d_A, *d_B, *d_C;
    half *d_A_half, *d_B_half;
    CHECK_CUDA(hipMalloc((void **) &d_A, m * k * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &d_B, k * n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &d_C, m * n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **) &d_A_half, m * k * sizeof(half)));
    CHECK_CUDA(hipMalloc((void **) &d_B_half, k * n * sizeof(half)));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice));

    // convert generated matrices to half precision
    convertFp32ToFp16<<<(m * k + 255) / 256, 256>>>(d_A_half, d_A, m * k);
    convertFp32ToFp16<<<(k * n + 255) / 256, 256>>>(d_B_half, d_B, k * n);

    // Constants for GEMM
    const float alpha = 1.0f;
    const float beta = 0.0f;


    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const char *device_name = prop.name;

    // add gpu name to the log
    // add sparsity to the log
    state.SetLabel("Arch:" + std::string(device_name));
    for (auto _: state) {
        float iteration_time_ms = 0.0f;

        MeasureTensorCorePerformance(m, n, k, d_A_half, d_B_half, d_C, alpha, beta, iteration_time_ms);

        state.SetIterationTime(iteration_time_ms);

    }
    // Final cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
}

// Define constants
const int M = 2048;
const int N = 2048;
const int K = 1024;
const int iters = 100;

// Register the function as a benchmark

BENCHMARK(BM_cuBLAS_CUDA)->Args({M, N, 32, 50})->Args({M, N, 64, 50})->Args({M, N, 128, 50})->Args({M, N, 512, 50})->
        ArgNames({"M", "N", "K", "Sparsity"})->Unit(benchmark::kMillisecond)->UseManualTime()->Iterations(iters);

BENCHMARK(BM_TensorCore_GEMM)->Args({M, N, 32, 50})->Args({M, N, 64, 50})->Args({M, N, 128, 50})->Args(
        {M, N, 512, 50})->
        ArgNames({"M", "N", "K", "Sparsity"})->Unit(benchmark::kMillisecond)->UseManualTime()->Iterations(iters);

BENCHMARK(BM_cuBLAS_TensorCore)->Args({M, N, 32, 50})->Args({M, N, 64, 50})->Args({M, N, 128, 50})->Args(
        {M, N, 512, 50})->
        ArgNames({"M", "N", "K", "Sparsity"})->Unit(benchmark::kMillisecond)->UseManualTime()->Iterations(iters);

BENCHMARK(BM_CUSPARSE_SPMM)->Args({M, N, 32, 50})->Args({M, N, 64, 50})->Args({M, N, 128, 50})->Args({M, N, 512, 50})
        ->Args({M, N, 32, 60})->Args({M, N, 64, 60})->Args({M, N, 128, 60})->Args({M, N, 512, 60})
        ->Args({M, N, 32, 70})->Args({M, N, 64, 70})->Args({M, N, 128, 70})->Args({M, N, 512, 70})
        ->Args({M, N, 32, 80})->Args({M, N, 64, 80})->Args({M, N, 128, 80})->Args({M, N, 512, 80})
        ->Args({M, N, 32, 90})->Args({M, N, 64, 90})->Args({M, N, 128, 90})->Args({M, N, 512, 90})
        ->Args({M, N, 32, 95})->Args({M, N, 64, 95})->Args({M, N, 128, 95})->Args({M, N, 512, 95})
        ->Args({M, N, 32, 99})->Args({M, N, 64, 99})->Args({M, N, 128, 99})->Args({M, N, 512, 99})
        ->ArgNames({"M", "N", "K", "Sparsity"})->Unit(benchmark::kMillisecond)->UseManualTime()->Iterations(iters);
// NOTE: manual time is reported in ms and as real time (not CPU time)



// Run the benchmark
//BENCHMARK_MAIN();
int main(int argc, char **argv) {
    benchmark::Initialize(&argc, argv);
    benchmark::RunSpecifiedBenchmarks();
    return 0;
}
